#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "../../gpu_utils.cu"



__global__ 
void vecAddKernel(float* A, float* B, float *C, int n){
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if(i < n){
    C[i] = A[i] + B[i];
  }
}

void vecAdd(float* A_h, float* B_h, float* C_h, int n){
  const int size = n * sizeof(float);
  float* A_d;
  float* B_d;
  float* C_d;

  gpuErrchk(hipMalloc((void**)&A_d, size));
  gpuErrchk(hipMalloc((void**)&B_d, size));
  gpuErrchk(hipMalloc((void**)&C_d, size));

  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
  hipMemcpy(C_d, C_h, size, hipMemcpyHostToDevice);

  int blockSize = 256;

  GpuTimer timer;
  timer.Start();
  vecAddKernel<<<(n+blockSize-1)/blockSize, blockSize>>>(A_d, B_d, C_d, n);
  timer.Stop();

  printf("Block Size %d, %f ms\n", blockSize, timer.Elapsed());

  hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

  hipFree(&A_d);
  hipFree(&B_d);
  hipFree(&C_d);
}

int main(){
  int n{};
  printf("Enter an array size: ");
  scanf("%d", &n);

  float* A = (float*)malloc(n * sizeof(float));
  float* B = (float*)malloc(n * sizeof(float));
  float* C = (float*)malloc(n * sizeof(float));

  for(int i = 0; i < n; ++i){
    A[i] = i;
    B[i] = i + 1;
  }
  vecAdd(A, B, C, n);

  free(A);
  free(B);
  free(C);
  return 0;
}